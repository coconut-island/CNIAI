#include "hip/hip_runtime.h"
//
// Created by abel on 23-2-26.
//

#include "cniai_cuda_kernel/preprocess.h"
#include "cniai_cuda_kernel/common.h"


namespace cniai {
namespace preprocess {


template<int c = 3, bool is_output_planar>
__global__ void rgb_resize_bilinear_pad_kernel(const uint8_t *src, uint8_t *dst,
                                           const int src_width, const int src_height,
                                           const int img_width, const int img_height,
                                           const int dst_width, const int dst_height,
                                           const int img_x, const int img_y,
                                           const int pad0, const int pad1, const int pad2,
                                           const float scale_x, const float scale_y) {
    const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height)
        return;

    float src_x = (dst_x - img_x) * scale_x;
    float src_y = (dst_y - img_y) * scale_y;

    bool is_in_img = img_y <= dst_y && dst_y < img_y + img_height && img_x <= dst_x && dst_x < img_x + img_width;
    for (int c_idx = 0; c_idx < c; c_idx++) {
        uint8_t out = 0;
        if (is_in_img) {
            const int x1      = __float2int_rd(src_x);
            const int y1      = __float2int_rd(src_y);
            const int x2      = x1 + 1;
            const int y2      = y1 + 1;
            const int x2_read = min(x2, src_width - 1);
            const int y2_read = min(y2, src_height - 1);

            uint8_t src_reg = src[y1 * src_width * c + x1 * c + c_idx];
            out = out + src_reg * ((x2 - src_x) * (y2 - src_y));

            src_reg = src[y1 * src_width * c + x2_read * c + c_idx];
            out = out + src_reg * ((src_x - x1) * (y2 - src_y));

            src_reg = src[y2_read * src_width * c + x1 * c + c_idx];
            out = out + src_reg * ((x2 - src_x) * (src_y - y1));

            src_reg = src[y2_read * src_width * c + x2_read * c + c_idx];
            out = out + src_reg * ((src_x - x1) * (src_y - y1));
        } else {
            out = c_idx == 0 ? pad0 : c_idx == 1 ? pad1 : pad2;
        }

        int dst_current_idx = is_output_planar ?
                              dst_width * dst_height * c_idx + dst_y * dst_width + dst_x :
                              dst_y * dst_width * c + dst_x * c + c_idx;

        dst[dst_current_idx] = out;

    }
}


void rgb_resize_bilinear_pad(const uint8_t *src, uint8_t *dst,
                                    const int src_width, const int src_height,
                                    const int img_width, const int img_height,
                                    const int dst_width, const int dst_height,
                                    const int img_x, const int img_y,
                                    const int pad0, const int pad1, const int pad2, hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dst_width + block.x - 1) / block.x, (dst_height + block.y - 1) / block.y);
    float scale_x = static_cast<float>(src_width) / img_width;
    float scale_y = static_cast<float>(src_height) / img_height;

    rgb_resize_bilinear_pad_kernel<3, false><<<grid, block, 0, cudaStream>>>(src, dst, src_width, src_height,
                                                                             img_width, img_height,
                                                                             dst_width, dst_height, img_x, img_y,
                                                                             pad0, pad1, pad2, scale_x, scale_y);
}


void rgb_resize_bilinear_pad_output_planar(const uint8_t *src, uint8_t *dst,
                                    const int src_width, const int src_height,
                                    const int img_width, const int img_height,
                                    const int dst_width, const int dst_height,
                                    const int img_x, const int img_y,
                                    const int pad0, const int pad1, const int pad2, hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dst_width + block.x - 1) / block.x, (dst_height + block.y - 1) / block.y);
    float scale_x = static_cast<float>(src_width) / img_width;
    float scale_y = static_cast<float>(src_height) / img_height;

    rgb_resize_bilinear_pad_kernel<3, true><<<grid, block, 0, cudaStream>>>(src, dst, src_width, src_height,
                                                                             img_width, img_height,
                                                                             dst_width, dst_height, img_x, img_y,
                                                                             pad0, pad1, pad2, scale_x, scale_y);
}

}}