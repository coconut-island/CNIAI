#include "hip/hip_runtime.h"
//
// Created by abel on 23-2-27.
//

#include "cniai_cuda_kernel/preprocess.h"
#include "cniai_cuda_kernel/common.h"


namespace cniai {
namespace preprocess {


template<int channel = 3, bool isOutputPlanar = false, bool isSwapRB = false>
__global__ void rgbResizeBilinearPadNormKernel(const uint8_t *src, float *dst,
                                               const int srcWidth, const int srcHeight,
                                               const int imgWidth, const int imgHeight,
                                               const int dstWidth, const int dstHeight,
                                               const int imgX, const int imgY,
                                               const int pad0, const int pad1, const int pad2,
                                               const float scale,
                                               const float mean0, const float mean1, const float mean2,
                                               const float std0, const float std1, const float std2,
                                               const float scaleX, const float scaleY) {
    const unsigned int dstX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int dstY = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstX >= dstWidth || dstY >= dstHeight)
        return;

    float srcX = static_cast<float>(dstX - imgX) * scaleX;
    float srcY = static_cast<float>(dstY - imgY) * scaleY;

    bool isInImg = imgY <= dstY && dstY < imgY + imgHeight && imgX <= dstX && dstX < imgX + imgWidth;
    for (int cIdx = 0; cIdx < channel; cIdx++) {
        float out = 0;
        if (isInImg) {
            const int x1 = __float2int_rd(srcX);
            const int y1 = __float2int_rd(srcY);
            const int x2 = x1 + 1;
            const int y2 = y1 + 1;
            const int x2Read = min(x2, srcWidth - 1);
            const int y2Read = min(y2, srcHeight - 1);

            uint8_t srcReg = src[y1 * srcWidth * channel + x1 * channel + cIdx];
            out = out + srcReg * ((x2 - srcX) * (y2 - srcY));

            srcReg = src[y1 * srcWidth * channel + x2Read * channel + cIdx];
            out = out + srcReg * ((srcX - x1) * (y2 - srcY));

            srcReg = src[y2Read * srcWidth * channel + x1 * channel + cIdx];
            out = out + srcReg * ((x2 - srcX) * (srcY - y1));

            srcReg = src[y2Read * srcWidth * channel + x2Read * channel + cIdx];
            out = out + srcReg * ((srcX - x1) * (srcY - y1));

            float mean = cIdx == 0 ? mean0 : cIdx == 1 ? mean1 : mean2;
            float std = cIdx == 0 ? std0 : cIdx == 1 ? std1 : std2;

            out = (out * scale - mean) * std;
        } else {
            out = cIdx == 0 ? pad0 : cIdx == 1 ? pad1 : pad2;
        }


        int curChannelIdx = cIdx;
        if (isSwapRB) {
            curChannelIdx = cIdx == 0 ? 2 : cIdx == 2 ? 0 : 1;
        }

        int dstCurrentIdx = isOutputPlanar ?
                              dstWidth * dstHeight * curChannelIdx + dstY * dstWidth + dstX :
                              dstY * dstWidth * channel + dstX * channel + curChannelIdx;

        dst[dstCurrentIdx] = out;

    }
}


void rgbResizeBilinearPadNorm(const uint8_t *src, float *dst,
                              const int srcWidth, const int srcHeight,
                              const int imgWidth, const int imgHeight,
                              const int dstWidth, const int dstHeight,
                              const int imgX, const int imgY,
                              const int pad0, const int pad1, const int pad2,
                              const float scale,
                              const float mean0, const float mean1, const float mean2,
                              const float std0, const float std1, const float std2,
                              hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dstWidth + block.x - 1) / block.x, (dstHeight + block.y - 1) / block.y);
    float scaleX = static_cast<float>(srcWidth) / static_cast<float>(imgWidth);
    float scaleY = static_cast<float>(srcHeight) / static_cast<float>(imgHeight);

    rgbResizeBilinearPadNormKernel<3, false><<<grid, block, 0, cudaStream>>>(
            src, dst,
            srcWidth, srcHeight,
            imgWidth, imgHeight,
            dstWidth, dstHeight,
            imgX, imgY,
            pad0, pad1, pad2,
            scale,
            mean0, mean1, mean2,
            std0, std1, std2,
            scaleX, scaleY);
}


void rgbResizeBilinearPadNormOutputPlanar(const uint8_t *src, float *dst,
                                          const int srcWidth, const int srcHeight,
                                          const int imgWidth, const int imgHeight,
                                          const int dstWidth, const int dstHeight,
                                          const int imgX, const int imgY,
                                          const int pad0, const int pad1, const int pad2,
                                          const float scale,
                                          const float mean0, const float mean1, const float mean2,
                                          const float std0, const float std1, const float std2,
                                          hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dstWidth + block.x - 1) / block.x, (dstHeight + block.y - 1) / block.y);
    float scaleX = static_cast<float>(srcWidth) / static_cast<float>(imgWidth);
    float scaleY = static_cast<float>(srcHeight) / static_cast<float>(imgHeight);

    rgbResizeBilinearPadNormKernel<3, true><<<grid, block, 0, cudaStream>>>(
            src, dst,
            srcWidth, srcHeight,
            imgWidth, imgHeight,
            dstWidth, dstHeight,
            imgX, imgY,
            pad0, pad1, pad2,
            scale,
            mean0, mean1, mean2,
            std0, std1, std2,
            scaleX, scaleY);
}


void rgbToBgrResizeBilinearPadNorm(const uint8_t *src, float *dst,
                                   const int srcWidth, const int srcHeight,
                                   const int imgWidth, const int imgHeight,
                                   const int dstWidth, const int dstHeight,
                                   const int imgX, const int imgY,
                                   const int pad0, const int pad1, const int pad2,
                                   const float scale,
                                   const float mean0, const float mean1, const float mean2,
                                   const float std0, const float std1, const float std2,
                                   hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dstWidth + block.x - 1) / block.x, (dstHeight + block.y - 1) / block.y);
    float scaleX = static_cast<float>(srcWidth) / static_cast<float>(imgWidth);
    float scaleY = static_cast<float>(srcHeight) / static_cast<float>(imgHeight);

    rgbResizeBilinearPadNormKernel<3, false, true><<<grid, block, 0, cudaStream>>>(
            src, dst,
            srcWidth, srcHeight,
            imgWidth, imgHeight,
            dstWidth, dstHeight,
            imgX, imgY,
            pad0, pad1, pad2,
            scale,
            mean0, mean1, mean2,
            std0, std1, std2,
            scaleX, scaleY);
}


void rgbToBgrResizeBilinearPadNormOutputPlanar(const uint8_t *src, float *dst,
                                               const int srcWidth, const int srcHeight,
                                               const int imgWidth, const int imgHeight,
                                               const int dstWidth, const int dstHeight,
                                               const int imgX, const int imgY,
                                               const int pad0, const int pad1, const int pad2,
                                               const float scale,
                                               const float mean0, const float mean1, const float mean2,
                                               const float std0, const float std1, const float std2,
                                               hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dstWidth + block.x - 1) / block.x, (dstHeight + block.y - 1) / block.y);
    float scaleX = static_cast<float>(srcWidth) / static_cast<float>(imgWidth);
    float scaleY = static_cast<float>(srcHeight) / static_cast<float>(imgHeight);

    rgbResizeBilinearPadNormKernel<3, true, true><<<grid, block, 0, cudaStream>>>(
            src, dst, srcWidth, srcHeight,
            imgWidth, imgHeight,
            dstWidth, dstHeight, imgX, imgY,
            pad0, pad1, pad2,
            scale,
            mean0, mean1, mean2,
            std0, std1, std2,
            scaleX, scaleY);
}


}
}